#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);

void printCudaInfo();

/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

#define index_type unsigned long

__global__ void first_loop(int *data, int N, index_type twod, index_type twod1, int numThreads) {
    index_type start = twod1 * (blockIdx.x * blockDim.x + threadIdx.x);
    for(index_type i = start; i + twod1 - 1 < N; i += twod1  * numThreads) {
            data[i + twod1 - 1] += data[i + twod - 1];
    }
}

__global__ void second_loop(int *data, int N, index_type twod, index_type twod1, int numThreads) {
    index_type start = twod1 * (blockIdx.x * blockDim.x + threadIdx.x);

    for(index_type i = start; i < N; i += twod1 * numThreads) {
            int t = data[i + twod - 1];
            data[i + twod - 1] = data[i + twod1 - 1];
            data[i + twod1 - 1] += t;
    }

}

void exclusive_scan(int *device_data, int length) {
    int threadsPerBlock = 32;
    int blockCnt = 256;
    int numThreads = threadsPerBlock * blockCnt;

    int N = nextPow2(length); // TODO round-up or something

    // upsweep phase.
    for (index_type twod = 1; twod < N; twod *= 2) {
        index_type twod1 = twod * 2;
        // TODO understand how this call differs when threads are in 2D or 3D
        first_loop<<<blockCnt, threadsPerBlock>>>(device_data, N, twod, twod1, numThreads);
    }

    int zero = 0;
    hipMemcpy(device_data + (N-1), &zero, sizeof(int), hipMemcpyHostToDevice);

    // downsweep phase.
    for (index_type twod = N / 2; twod >= 1; twod /= 2) {
        index_type twod1 = twod * 2;
        second_loop<<<blockCnt, threadsPerBlock>>>(device_data, N, twod, twod1, numThreads);
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int *inarray, int *end, int *resultarray) {
    int *device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);

    // TODO delete
//    printf("\n100 first Results:\n");
//    for (int i = 0; i < 100; i++) {
//        printf("%d ", resultarray[i]);
//    }
//    printf("\n");

    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}



int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    return 0;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);

        printf("deviceProps.maxThreadsPerBlockk: %d\n", deviceProps.maxThreadsPerBlock);
        printf("deviceProp.maxThreadsPerMultiProcessor: %d\n", deviceProps
        .maxThreadsPerMultiProcessor);
    }
    printf("---------------------------------------------------------\n");
}
